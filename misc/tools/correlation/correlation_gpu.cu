/*
 * correlation_gpu_kernel.cu
 * 
 * Copyright (C) 2015, Achim Lösch <achim.loesch@upb.de>, Christoph Knorr <cknorr@mail.uni-paderborn.de>
 * All rights reserved.
 * 
 * This software may be modified and distributed under the terms
 * of the BSD license. See the LICENSE file for details.
 * 
 * encoding: UTF-8
 * tab size: 4
 * 
 * author: Achim Lösch (achim.loesch@upb.de)
 * created: 2/04/15
 * version: 0.3.1 - add correlation example application
 */

#include "correlation_gpu.h"

#include "correlation_gpu_kernel.h"

#include "settings.h"
#include "time_watch.h"

#include <stdint.h>

#ifdef ORIGINAL

void exec_correlation_app(unsigned int granularity, DATA *stddev, DATA *mean, DATA *symmat, DATA *data) {
	DATA float_n		= (DATA)N_CORRELATION;
	
	/* GPU pointers */
	DATA *stddev_gpu	= NULL;
	DATA *mean_gpu		= NULL;
	DATA *symmat_gpu	= NULL;
	DATA *data_gpu		= NULL;
	DATA *float_n_gpu	= NULL;
	
	int cuda_device_nr	= 0;
	
	/*
	 * INIT
	 */
	CUDACALL( hipSetDevice(cuda_device_nr) );
	
	CUDACALL( hipMalloc((void**)&symmat_gpu, SIZE_MATRIX) );
	CUDACALL( hipMalloc((void**)&data_gpu, SIZE_MATRIX) );
	
	CUDACALL( hipMalloc((void**)&mean_gpu, SIZE_VECTOR) );
	CUDACALL( hipMalloc((void**)&stddev_gpu, SIZE_VECTOR) );
	CUDACALL( hipMalloc((void**)&float_n_gpu, SIZE_SCALAR) );
	
	#ifdef ENERGY_LOOPS
	uint32_t e;
	for (e=0; e<E_CORRELATION; ++e) {
		CUDACALL( hipMemcpyAsync(symmat_gpu, symmat, SIZE_MATRIX, hipMemcpyHostToDevice) );
		CUDACALL( hipMemcpyAsync(data_gpu, data, SIZE_MATRIX, hipMemcpyHostToDevice) );
		
		CUDACALL( hipMemcpyAsync(mean_gpu, mean, SIZE_VECTOR, hipMemcpyHostToDevice) );
		CUDACALL( hipMemcpyAsync(stddev_gpu, stddev, SIZE_VECTOR, hipMemcpyHostToDevice) );
		CUDACALL( hipMemcpyAsync(float_n_gpu, &float_n, SIZE_SCALAR, hipMemcpyHostToDevice) );
	}
	#else /* NO ENERGY_LOOPS */
	CUDACALL( hipMemcpyAsync(symmat_gpu, symmat, SIZE_MATRIX, hipMemcpyHostToDevice) );
	CUDACALL( hipMemcpyAsync(data_gpu, data, SIZE_MATRIX, hipMemcpyHostToDevice) );
	
	CUDACALL( hipMemcpyAsync(mean_gpu, mean, SIZE_VECTOR, hipMemcpyHostToDevice) );
	CUDACALL( hipMemcpyAsync(stddev_gpu, stddev, SIZE_VECTOR, hipMemcpyHostToDevice) );
	CUDACALL( hipMemcpyAsync(float_n_gpu, &float_n, SIZE_SCALAR, hipMemcpyHostToDevice) );
	#endif /* ENERGY_LOOPS */
	
	/*
	 * COMPUTE
	 */
	dim3 block(NTHREADSX, NTHREADSY);
	dim3 grid(NBLOCKSX, NBLOCKSY);
	
	correlation_calc_data_gpuOPT <<<grid, block,0>>> ((DATA(*)[N_CORRELATION])data_gpu, mean_gpu, stddev_gpu, float_n_gpu);
	CHECKCUDAERROR("correlation_calc_data_gpuOPT launch failed");
	
	int cur = 0;
	for (cur=0; cur<N_CORRELATION; cur += granularity) {
	#ifdef ENERGY_LOOPS
		for (e=0; e<E_CORRELATION; ++e) {
			dim3 grid2(NBLOCKSX, NBLOCKSY);
			dim3 block2(NTHREADSX, NTHREADSY, NTHREADSZ);
			
			correlation_calc_gpuOPT1 <<<grid2, block2,0>>> (cur, granularity, (DATA(*)[N_CORRELATION])data_gpu, (DATA(*)[N_CORRELATION])symmat_gpu);
			CHECKCUDAERROR("correlation_calc_gpuOPT1 launch failed");
			
			CUDACALL( hipDeviceSynchronize());
			
			correlation_calc_gpuOPT2 <<<grid2, block2,0>>> (cur, granularity, (DATA(*)[N_CORRELATION])data_gpu, (DATA(*)[N_CORRELATION])symmat_gpu);
			CHECKCUDAERROR("correlation_calc_gpuOPT2 launch failed");
			
			CUDACALL( hipDeviceSynchronize());
			
			correlation_calc_gpuOPT3 <<<grid2, block2,0>>> (cur, granularity, (DATA(*)[N_CORRELATION])data_gpu, (DATA(*)[N_CORRELATION])symmat_gpu);
			CHECKCUDAERROR("correlation_calc_gpuOPT3 launch failed");
			
			CUDACALL( hipDeviceSynchronize());
		}
	#else /* NO ENERGY_LOOPS */
		dim3 grid2(NBLOCKSX, NBLOCKSY);
		dim3 block2(NTHREADSX, NTHREADSY, NTHREADSZ);
		
		correlation_calc_gpuOPT1 <<<grid2, block2,0>>> (cur, granularity, (DATA(*)[N_CORRELATION])data_gpu, (DATA(*)[N_CORRELATION])symmat_gpu);
		CHECKCUDAERROR("correlation_calc_gpuOPT1 launch failed");
		
		CUDACALL( hipDeviceSynchronize());
		
		correlation_calc_gpuOPT2 <<<grid2, block2,0>>> (cur, granularity, (DATA(*)[N_CORRELATION])data_gpu, (DATA(*)[N_CORRELATION])symmat_gpu);
		CHECKCUDAERROR("correlation_calc_gpuOPT2 launch failed");
		
		CUDACALL( hipDeviceSynchronize());
		
		correlation_calc_gpuOPT3 <<<grid2, block2,0>>> (cur, granularity, (DATA(*)[N_CORRELATION])data_gpu, (DATA(*)[N_CORRELATION])symmat_gpu);
		CHECKCUDAERROR("correlation_calc_gpuOPT3 launch failed");
		
		CUDACALL( hipDeviceSynchronize());
	#endif /* ENERGY_LOOPS */
	}
	
	/*
	 * FREE
	 */
	#ifdef ENERGY_LOOPS
	for (e=0; e<E_CORRELATION; ++e) {
		CUDACALL( hipMemcpyAsync(data, data_gpu, SIZE_MATRIX, hipMemcpyDeviceToHost) );
		CUDACALL( hipMemcpyAsync(symmat, symmat_gpu, SIZE_MATRIX, hipMemcpyDeviceToHost) );
		
		CUDACALL( hipDeviceSynchronize());
	}
	#else /* NO ENERGY_LOOPS */
	CUDACALL( hipMemcpyAsync(data, data_gpu, SIZE_MATRIX, hipMemcpyDeviceToHost) );
	CUDACALL( hipMemcpyAsync(symmat, symmat_gpu, SIZE_MATRIX, hipMemcpyDeviceToHost) );
	
	CUDACALL( hipDeviceSynchronize());
	#endif /* ENERGY_LOOPS */
	
	CUDACALL( hipFree(mean_gpu) );
	CUDACALL( hipFree(stddev_gpu) );
	CUDACALL( hipFree(float_n_gpu) );
	CUDACALL( hipFree(data_gpu) );
	CUDACALL( hipFree(symmat_gpu) );
}

#else /* PROGRAMMING PATTERN */

static void exec_correlation_app_init(WORKER *worker);
static int exec_correlation_app_compute(WORKER *worker);
static void exec_correlation_app_free(WORKER *worker);

static void exec_correlation_app_init(WORKER *worker) {
	CUDACALL( hipSetDevice(worker->cuda_device_nr) );

	CUDACALL( hipMalloc((void**)&(worker->symmat_gpu), SIZE_MATRIX) );
	CUDACALL( hipMalloc((void**)&(worker->data_gpu), SIZE_MATRIX) );
	if (worker->cur == 0) {
		CUDACALL( hipMalloc((void**)&(worker->mean_gpu), SIZE_VECTOR) );
		CUDACALL( hipMalloc((void**)&(worker->stddev_gpu), SIZE_VECTOR) );
		CUDACALL( hipMalloc((void**)&(worker->float_n_gpu), SIZE_SCALAR) );
	}
	
	#ifdef ENERGY_LOOPS
		uint32_t e;
		for (e=0; e<E_CORRELATION; ++e) {
			CUDACALL( hipMemcpyAsync(worker->symmat_gpu, worker->symmat, SIZE_MATRIX, hipMemcpyHostToDevice) );
			CUDACALL( hipMemcpyAsync(worker->data_gpu, worker->data, SIZE_MATRIX, hipMemcpyHostToDevice) );
			if (worker->cur == 0) {
				CUDACALL( hipMemcpyAsync(worker->mean_gpu, worker->mean, SIZE_VECTOR, hipMemcpyHostToDevice) );
				CUDACALL( hipMemcpyAsync(worker->stddev_gpu, worker->stddev, SIZE_VECTOR, hipMemcpyHostToDevice) );
				CUDACALL( hipMemcpyAsync(worker->float_n_gpu, &(worker->float_n), SIZE_SCALAR, hipMemcpyHostToDevice) );
			}
		}
	#else /* NO ENERGY_LOOPS */
		CUDACALL( hipMemcpyAsync(worker->symmat_gpu, worker->symmat, SIZE_MATRIX, hipMemcpyHostToDevice) );
		CUDACALL( hipMemcpyAsync(worker->data_gpu, worker->data, SIZE_MATRIX, hipMemcpyHostToDevice) );
		if (worker->cur == 0) {
			CUDACALL( hipMemcpyAsync(worker->mean_gpu, worker->mean, SIZE_VECTOR, hipMemcpyHostToDevice) );
			CUDACALL( hipMemcpyAsync(worker->stddev_gpu, worker->stddev, SIZE_VECTOR, hipMemcpyHostToDevice) );
			CUDACALL( hipMemcpyAsync(worker->float_n_gpu, &(worker->float_n), SIZE_SCALAR, hipMemcpyHostToDevice) );
		}
	#endif /* ENERGY_LOOPS */
}

static int exec_correlation_app_compute(WORKER *worker) {
	if (worker->cur == 0) {
		dim3 block(NTHREADSX, NTHREADSY);
		dim3 grid(NBLOCKSX, NBLOCKSY);
		
		correlation_calc_data_gpuOPT <<<grid, block,0>>> ((DATA(*)[N_CORRELATION])(worker->data_gpu), worker->mean_gpu, worker->stddev_gpu, worker->float_n_gpu);
		CHECKCUDAERROR("correlation_calc_data_gpuOPT launch failed");
		
		CUDACALL( hipFree(worker->mean_gpu) );
		CUDACALL( hipFree(worker->stddev_gpu) );
		CUDACALL( hipFree(worker->float_n_gpu) );
	}
	
	#ifdef ENERGY_LOOPS
		uint32_t e;
		for (e=0; e<E_CORRELATION; ++e) {
			dim3 grid2(NBLOCKSX, NBLOCKSY);
			dim3 block2(NTHREADSX, NTHREADSY, NTHREADSZ);
			
			correlation_calc_gpuOPT1 <<<grid2, block2,0>>> (worker->cur, worker->granularity, (DATA(*)[N_CORRELATION])(worker->data_gpu), (DATA(*)[N_CORRELATION])(worker->symmat_gpu));
			CHECKCUDAERROR("correlation_calc_gpuOPT1 launch failed");
			
			CUDACALL( hipDeviceSynchronize());
			
			correlation_calc_gpuOPT2 <<<grid2, block2,0>>> (worker->cur, worker->granularity, (DATA(*)[N_CORRELATION])(worker->data_gpu), (DATA(*)[N_CORRELATION])(worker->symmat_gpu));
			CHECKCUDAERROR("correlation_calc_gpuOPT2 launch failed");
			
			CUDACALL( hipDeviceSynchronize());
			
			correlation_calc_gpuOPT3 <<<grid2, block2,0>>> (worker->cur, worker->granularity, (DATA(*)[N_CORRELATION])(worker->data_gpu), (DATA(*)[N_CORRELATION])(worker->symmat_gpu));
			CHECKCUDAERROR("correlation_calc_gpuOPT3 launch failed");
			
			CUDACALL( hipDeviceSynchronize());
		}
	#else /* NO ENERGY_LOOPS */
		dim3 grid2(NBLOCKSX, NBLOCKSY);
		dim3 block2(NTHREADSX, NTHREADSY, NTHREADSZ);
		
		correlation_calc_gpuOPT1 <<<grid2, block2,0>>> (worker->cur, worker->granularity, (DATA(*)[N_CORRELATION])(worker->data_gpu), (DATA(*)[N_CORRELATION])(worker->symmat_gpu));
		CHECKCUDAERROR("correlation_calc_gpuOPT1 launch failed");
		
		CUDACALL( hipDeviceSynchronize());
		
		correlation_calc_gpuOPT2 <<<grid2, block2,0>>> (worker->cur, worker->granularity, (DATA(*)[N_CORRELATION])(worker->data_gpu), (DATA(*)[N_CORRELATION])(worker->symmat_gpu));
		CHECKCUDAERROR("correlation_calc_gpuOPT2 launch failed");
		
		CUDACALL( hipDeviceSynchronize());
		
		correlation_calc_gpuOPT3 <<<grid2, block2,0>>> (worker->cur, worker->granularity, (DATA(*)[N_CORRELATION])(worker->data_gpu), (DATA(*)[N_CORRELATION])(worker->symmat_gpu));
		CHECKCUDAERROR("correlation_calc_gpuOPT3 launch failed");
		
		CUDACALL( hipDeviceSynchronize());
	#endif /* ENERGY_LOOPS */
	
	worker->cur += worker->granularity;
	
	if((worker->cur)>=N_CORRELATION) {
		return 1;
	}
	
	return 0;
}

static void exec_correlation_app_free(WORKER *worker) {
	#ifdef ENERGY_LOOPS
		uint32_t e;
		for (e=0; e<E_CORRELATION; ++e) {
			CUDACALL( hipMemcpyAsync(worker->data, worker->data_gpu, SIZE_MATRIX, hipMemcpyDeviceToHost) );
			CUDACALL( hipMemcpyAsync(worker->symmat, worker->symmat_gpu, SIZE_MATRIX, hipMemcpyDeviceToHost) );
			
			CUDACALL( hipDeviceSynchronize());
		}
	#else /* NO ENERGY_LOOPS */
		CUDACALL( hipMemcpyAsync(worker->data, worker->data_gpu, SIZE_MATRIX, hipMemcpyDeviceToHost) );
		CUDACALL( hipMemcpyAsync(worker->symmat, worker->symmat_gpu, SIZE_MATRIX, hipMemcpyDeviceToHost) );
		
		CUDACALL( hipDeviceSynchronize());
	#endif /* ENERGY_LOOPS */
	
	CUDACALL( hipFree(worker->data_gpu) );
	CUDACALL( hipFree(worker->symmat_gpu) );
}

void exec_correlation_app(WORKER *worker) {
	int break_loop = 1;
	
	exec_correlation_app_init(worker);
	
	do {
		break_loop = exec_correlation_app_compute(worker);
	} while (!break_loop);
	
	exec_correlation_app_free(worker);
	
#if 0
	int i, j;
	for (i=0; i<N_CORRELATION; ++i) {
		for (j=0; j<N_CORRELATION; ++j) {
			printf("%.5f|", worker->data[i*N_CORRELATION+j]);
		}
		printf("\n");
	}
	
	for (i=0; i<N_CORRELATION; ++i) {
		for (j=0; j<N_CORRELATION; ++j) {
			printf("%.5f|", worker->symmat[i*N_CORRELATION+j]);
		}
		printf("\n");
	}
#endif
}

#endif /* ORIGINAL */
