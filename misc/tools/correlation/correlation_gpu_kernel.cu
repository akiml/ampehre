#include "hip/hip_runtime.h"
/*
 * correlation_gpu_kernel.cu
 * 
 * Copyright (C) 2015, Achim Lösch <achim.loesch@upb.de>, Christoph Knorr <cknorr@mail.uni-paderborn.de>
 * All rights reserved.
 * 
 * This software may be modified and distributed under the terms
 * of the BSD license. See the LICENSE file for details.
 * 
 * encoding: UTF-8
 * tab size: 4
 * 
 * author: Achim Lösch (achim.loesch@upb.de)
 * created: 2/04/15
 * version: 0.3.1 - add correlation example application
 */

#include "correlation_gpu_kernel.h"

__global__ void correlation_calc_gpuOPT1(unsigned int cur, unsigned int gran, float (*data_d)[N_CORRELATION], float (*symmat_d)[N_CORRELATION]) {
	int BLOCKY = blockIdx.y;
	int BLOCKX = blockIdx.x;
	int THREADY = threadIdx.y;
	int THREADX = threadIdx.x;
	
	int t2, t3, t5,t6;
	
	if ((cur <= N_CORRELATION-2) && (gran >= 1)) {
		for (t2=ceild(cur,16)+BLOCKY;t2<=min(floord(N_CORRELATION-2,16),floord(cur+gran-1,16));t2+=16) {
			for (t3=(max(ceild(cur,16),t2))+BLOCKX;t3<=floord(N_CORRELATION-1,16);t3+=16) {
				for (t5=(max(cur,16*t2))+THREADY;t5<=min(min(min(N_CORRELATION-2,16*t2+15),16*t3+14),cur+gran-1);t5+=16) {
					for (t6=(max(16*t3,t5+1))+THREADX;t6<=min(N_CORRELATION-1,16*t3+15);t6+=16) {
						symmat_d[t5][t6]=0.0;;
					}
				}
			}
		}
		for (t2=ceild(cur,16)+BLOCKY;t2<=min(floord(N_CORRELATION-2,16),floord(cur+gran-1,16));t2+=16) {
			for (t5=max(cur,16*t2);t5<=min(min(N_CORRELATION-2,16*t2+15),cur+gran-1);t5++) {
				symmat_d[t5][t5]=1.0;;
			}
		}
	}
}

__global__ void correlation_calc_gpuOPT2(unsigned int cur, unsigned int gran, float (*data)[N_CORRELATION], float (*symmat)[N_CORRELATION]) {
	int BLOCKY = blockIdx.y;
	int BLOCKX = blockIdx.x;
	int THREADY = threadIdx.y;
	int THREADX = threadIdx.x;
	
	int t1, t2, t3, t4, t5, t6;
	
	if ((N_CORRELATION >= 1) && (cur <= N_CORRELATION-1)) {
		for (t1=ceild(cur,16)+BLOCKY;t1<=min(floord(N_CORRELATION-2,16),floord(cur+gran-1,16));t1+=16) {
			for (t2=(max(ceild(cur,16),t1))+BLOCKX;t2<=floord(N_CORRELATION-1,16);t2+=16) {
				for (t3=0;t3<=floord(N_CORRELATION-1,16);t3++) {
					for (t4=(max(cur,16*t1))+THREADY;t4<=min(min(min(N_CORRELATION-2,cur+gran-1),16*t1+15),16*t2+14);t4+=16) {
						for (t5=(max(16*t2,t4+1))+THREADX;t5<=min(N_CORRELATION-1,16*t2+15);t5+=16) {
							for (t6=max(1,16*t3);t6<=min(N_CORRELATION-1,16*t3+15);t6++) {
								symmat[t4][t5]+=(data[t6][t4]*data[t6][t5]);;
							}
						}
					}
				}
			}
		}
	}
}

__global__ void correlation_calc_gpuOPT3(unsigned int cur, unsigned int gran, float (*data)[N_CORRELATION], float (*symmat)[N_CORRELATION]) {
	int BLOCKY = blockIdx.y;
	int BLOCKX = blockIdx.x;
	int THREADY = threadIdx.y;
	int THREADX = threadIdx.x;
	
	int t2, t3,t5,t6;
	
	if ((cur < N_CORRELATION)) {
		for (t2=ceild(cur,16)+BLOCKY;t2<=min(floord(N_CORRELATION-2,16),floord(cur+gran-1,16));t2+=16) {
			for (t3=(max(ceild(cur,16),t2))+BLOCKX;t3<=floord(N_CORRELATION-1,16);t3+=16) {
				for (t5=(max(cur,16*t2))+THREADY;t5<=min(min(min(N_CORRELATION-2,16*t2+15),16*t3+14),cur+gran-1);t5+=16) {
					for (t6=(max(16*t3,t5+1))+THREADX;t6<=min(N_CORRELATION-1,16*t3+15);t6+=16) {
						symmat[t6][t5]=symmat[t5][t6];;
					}
				}
			}
		}
	}
}

extern "C" void init_correlation_gpuOPT(float (*data)[N_CORRELATION], float (*symmat)[N_CORRELATION], float** data_d, float** symmat_d, int device_num) {
	CUDACALL( hipSetDevice(device_num) );
	
	CUDACALL( hipMalloc((void**)data_d, (N_CORRELATION)*(N_CORRELATION)*sizeof(float)) );
	CUDACALL( hipMalloc((void**)symmat_d, (N_CORRELATION)*(N_CORRELATION)*sizeof(float)) );
	
	CUDACALL( hipMemcpyAsync(*data_d, data, (N_CORRELATION)*(N_CORRELATION)*sizeof(float), hipMemcpyHostToDevice) ); //NONBLOCKING
	CUDACALL( hipMemcpyAsync(*symmat_d, symmat, (N_CORRELATION)*(N_CORRELATION)*sizeof(float), hipMemcpyHostToDevice) ); //NONBLOCKING
	
	CUDACALL( hipDeviceSynchronize());
}

extern "C" void free_correlation_gpuOPT(float (*data)[N_CORRELATION], float (*symmat)[N_CORRELATION], float* data_d, float* symmat_d) {
	CUDACALL( hipMemcpyAsync(data, data_d, (N_CORRELATION)*(N_CORRELATION)*sizeof(float), hipMemcpyDeviceToHost) ); //NONBLOCKING
	CUDACALL( hipMemcpyAsync(symmat, symmat_d, (N_CORRELATION)*(N_CORRELATION)*sizeof(float), hipMemcpyDeviceToHost) ); //NONBLOCKING
	
	CUDACALL( hipDeviceSynchronize());
	
	CUDACALL( hipFree(data_d) );
	CUDACALL( hipFree(symmat_d) );
}

extern "C" void launch_correlation_gpuOPT(unsigned int cur, unsigned int gran, float* data_d, float* symmat_d) {
	dim3 grid(NBLOCKSX, NBLOCKSY);
	dim3 block(NTHREADSX, NTHREADSY, NTHREADSZ);
	
	correlation_calc_gpuOPT1 <<<grid,block,0>>> (cur, gran, (float(*)[N_CORRELATION])data_d, (float(*)[N_CORRELATION])symmat_d);
	CHECKCUDAERROR("correlation_calc_gpuOPT1 launch failed");
	
	correlation_calc_gpuOPT2 <<<grid,block,0>>> (cur, gran, (float(*)[N_CORRELATION])data_d, (float(*)[N_CORRELATION])symmat_d);
	CHECKCUDAERROR("correlation_calc_gpuOPT2 launch failed");
	
	correlation_calc_gpuOPT3 <<<grid,block,0>>> (cur, gran, (float(*)[N_CORRELATION])data_d, (float(*)[N_CORRELATION])symmat_d);
	CHECKCUDAERROR("correlation_calc_gpuOPT3 launch failed");
}

__global__ void correlation_calc_data_gpuOPT(float (*data_d)[N_CORRELATION], float *mean_d, float *stddev_d, float* float_n_d) {
	int BLOCKY = blockIdx.y;
	int BLOCKX = blockIdx.x;
	int THREADY = threadIdx.y;
	int THREADX = threadIdx.x;
	
	int t2, t3, t4, t5;
	
	if (N_CORRELATION >= 1) {
		for (t2=0+BLOCKY;t2<=floord(N_CORRELATION-1,16);t2+=16) {
			for (t3=0+BLOCKX;t3<=floord(N_CORRELATION-1,16);t3+=16) {
				for (t4=16*t2+THREADY;t4<=min(N_CORRELATION-1,16*t2+15);t4+=16) {
					for (t5=16*t3+THREADX;t5<=min(N_CORRELATION-1,16*t3+15);t5+=16) {
						data_d[t4][t5]-=mean_d[t5];;
						data_d[t4][t5]/=sqrt(*float_n_d)*stddev_d[t5];;
					}
				}
			}
		}
	}
}

extern "C" void launch_correlation_data_gpuOPT(float (*data)[N_CORRELATION], float *mean, float *stddev, float float_n, int device_num) {
	CUDACALL( hipSetDevice(device_num) );
	
	float* data_d;
	float* mean_d;
	float* stddev_d;
	float* float_n_d;
	
	CUDACALL( hipMalloc((void**)&data_d, (N_CORRELATION)*(N_CORRELATION)*sizeof(float)) );
	CUDACALL( hipMalloc((void**)&mean_d, (N_CORRELATION)*sizeof(float)) );
	CUDACALL( hipMalloc((void**)&stddev_d, (N_CORRELATION)*sizeof(float)) );
	CUDACALL( hipMalloc((void**)&float_n_d, sizeof(float)) );
	
	CUDACALL( hipMemcpyAsync(data_d, data, (N_CORRELATION)*(N_CORRELATION)*sizeof(float), hipMemcpyHostToDevice) ); //NONBLOCKING
	CUDACALL( hipMemcpyAsync(mean_d, mean, (N_CORRELATION)*sizeof(float), hipMemcpyHostToDevice) ); //NONBLOCKING
	CUDACALL( hipMemcpyAsync(stddev_d, stddev, (N_CORRELATION)*sizeof(float), hipMemcpyHostToDevice) ); //NONBLOCKING
	CUDACALL( hipMemcpyAsync(float_n_d, &float_n, sizeof(float), hipMemcpyHostToDevice) ); //NONBLOCKING
	
	dim3 block(NTHREADSX, NTHREADSY);
	dim3 grid(NBLOCKSX, NBLOCKSY);
	
	correlation_calc_data_gpuOPT <<<grid, block,0>>> ((float(*)[N_CORRELATION])data_d, mean_d, stddev_d, float_n_d);
	CHECKCUDAERROR("heat_calc_data_gpuOPT launch failed");
	
	CUDACALL( hipMemcpyAsync(data, data_d, (N_CORRELATION)*(N_CORRELATION)*sizeof(float), hipMemcpyDeviceToHost) ); //NONBLOCKING
	CUDACALL( hipMemcpyAsync(mean, mean_d, (N_CORRELATION)*sizeof(float), hipMemcpyDeviceToHost) ); //NONBLOCKING
	CUDACALL( hipMemcpyAsync(stddev, stddev_d, (N_CORRELATION)*sizeof(float), hipMemcpyDeviceToHost) ); //NONBLOCKING
	CUDACALL( hipMemcpyAsync(&float_n, float_n_d, sizeof(float), hipMemcpyDeviceToHost) ); //NONBLOCKING
	
	CUDACALL( hipFree(data_d) );
	CUDACALL( hipFree(mean_d) );
	CUDACALL( hipFree(stddev_d) );
	CUDACALL( hipFree(float_n_d) );
}
